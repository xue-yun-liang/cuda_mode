#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "error.cuh"

#ifdef DUSE_DP
typedef double real;
#else
typedef float real;
#endif

const int TILE_DIM = 32;


// version 0: copy
__global__ void copy(const float *matrix_A, float* matrix_B, const int N){
    const int nx = threadIdx.x + blockIdx.x * TILE_DIM;
    const int ny = threadIdx.y + blockIdx.y * TILE_DIM;
    const int idx = ny * N + nx;
    if (nx < N && ny < N) {
        matrix_B[idx] = matrix_A[idx];
    }
}

// version 1: coalesced read, but uncoalesced write
__global__ void transpose_coalesced_read(float* matrix, float* matrix_t, const int N){
    const int nx = threadIdx.x + blockIdx.x * TILE_DIM;
    const int ny = threadIdx.y + blockIdx.y * TILE_DIM;
    if (nx < N && ny < N) {
        matrix_t[ny * N + nx] = matrix[nx * N + ny];
    }
}


// version 2: coalesced write, but uncoalesced read
// As the uncoalesced read is trabsform by complier to 
// matrix_t[nx * N + ny] = __ldg(&matrix[ny * N + nx]);
__global__ void transpose_coalesced_write(float* matrix, float* matrix_t, const int N){
    const int nx = threadIdx.x + blockIdx.x * TILE_DIM;
    const int ny = threadIdx.y + blockIdx.y * TILE_DIM;
    if (nx < N && ny < N) {
        matrix_t[nx * N + ny] = matrix[ny * N + nx];
    }
}


// version 3: using share memory,but with bank conflict
__global__ void transpose_share_v3(float* matrix, float* matrix_t, const int N){
    __shared__ real S[TILE_DIM][TILE_DIM];
    int bx = blockIdx.x * TILE_DIM;
    int by = blockIdx.y * TILE_DIM;

    int nx1 = bx + threadIdx.x;
    int ny1 = by + threadIdx.y;
    
    // copy date from global memo to share memo
    if(nx1 < N && ny1 < N){
        S[threadIdx.x][threadIdx.y] = matrix[ny1 * N + nx1];
    }
    __syncthreads();

    int nx2 = bx + threadIdx.x;
    int ny2 = by + threadIdx.y;
    if (nx2 < N && ny2 < N){
        matrix_t[nx2 * N + ny2] = S[threadIdx.x][threadIdx.y];
    }
}


// version 4: using share memory,without bank conflict
__global__ void transpose_share_v4(float* matrix, float* matrix_t, const int N){
    __shared__ real S[TILE_DIM][TILE_DIM];
    int bx = blockIdx.x * TILE_DIM;
    int by = blockIdx.y * TILE_DIM;

    int nx1 = bx + threadIdx.x;
    int ny1 = by + threadIdx.y;
    
    // copy date from global memo to share memo
    if(nx1 < N && ny1 < N){
        S[threadIdx.x][threadIdx.y] = matrix[ny1 * N + nx1];
    }
    __syncthreads();

    int nx2 = bx + threadIdx.x;
    int ny2 = by + threadIdx.y;
    if (nx2 < N && ny2 < N){
        matrix_t[nx2 * N + ny2] = S[threadIdx.x][threadIdx.y];
    }
}


// version 5: using share memory, without bank conflict, by alloc more share memory
__global__ void transpose_share(float* matrix, float* matrix_t, const int N){
    __shared__ real S[TILE_DIM][TILE_DIM + 1];
    int bx = blockIdx.x * TILE_DIM;
    int by = blockIdx.y * TILE_DIM;

    int nx1 = bx + threadIdx.x;
    int ny1 = by + threadIdx.y;
    
    // copy date from global memo to share memo
    if(nx1 < N && ny1 < N){
        S[threadIdx.x][threadIdx.y] = matrix[ny1 * N + nx1];
    }
    __syncthreads();

    int nx2 = bx + threadIdx.x;
    int ny2 = by + threadIdx.y;
    if (nx2 < N && ny2 < N){
        matrix_t[nx2 * N + ny2] = S[threadIdx.x][threadIdx.y];
    }
}


int main(){
    const int N = 32;              // the length of matric
    const int grid_size_x = (N + TILE_DIM - 1) / TILE_DIM;
    const int grid_size_y = grid_size_x;
    const dim3 block_size(TILE_DIM, TILE_DIM);
    const dim3 grid_size(grid_size_x, grid_size_y);

    // Allocate host memory
    size_t size = N * N * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);

    // Initialize host matrix A
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(i);
    }

    // Allocate device memory
    float *d_A, *d_B;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // exec the func
    transpose_share<<<grid_size, block_size>>>(d_A, d_B, N);

    // Copy the result from device to host
    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

    // Print the matrix B
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%2.0f ", h_B[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);

    // Free host memory
    free(h_A);
    free(h_B);
    
    return 0;
}

