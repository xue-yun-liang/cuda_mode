#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include "../include/error.cuh"
#include "../include/kernel.cuh"

// Dot Product
// grid(N/128), block(128)
// a: Nx1, b: Nx1, y=sum(elementwise_mul(a,b))
template<const int NUM_THREADS>
__global__ void dot(float* a, float* b, float* y, int N) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * NUM_THREADS + tid;
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  __shared__ float reduce_smem[NUM_WARPS];

  // keep the data in register is enougth for warp operaion.
  float prod = (idx < N) ? a[idx] * b[idx] : 0.0f;
  int warp = tid / WARP_SIZE;
  int lane = tid % WARP_SIZE;
  // perform warp sync reduce.
  prod = warp_reduce_sum<WARP_SIZE>(prod);
  // warp leaders store the data to shared memory.
  if (lane == 0) reduce_smem[warp] = prod;
  __syncthreads(); // make sure the data is in shared memory.
  // the first warp compute the final sum.
  prod = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f;
  if (warp == 0) prod = warp_reduce_sum<NUM_WARPS>(prod);
  if (tid == 0) atomicAdd(y, prod);
}

// Dot Product + Vec4
// grid(N/128), block(128/4)
// a: Nx1, b: Nx1, y=sum(elementwise_mul(a,b))
template<const int NUM_THREADS>
__global__ void dot_vec4(float* a, float* b, float* y, int N) {
  int tid = threadIdx.x;
  int idx = (blockIdx.x * NUM_THREADS + tid) * 4;
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  __shared__ float reduce_smem[NUM_WARPS];

  float4 reg_a = FLOAT4(a[idx]);
  float4 reg_b = FLOAT4(b[idx]);
  float prod = (idx < N) ? (reg_a.x * reg_b.x + reg_a.y * reg_b.y 
                          + reg_a.z * reg_b.z + reg_a.w * reg_b.w) : 0.0f;
  int warp = tid / WARP_SIZE;
  int lane = tid % WARP_SIZE;
  // perform warp sync reduce.
  prod = warp_reduce_sum<WARP_SIZE>(prod);
  // warp leaders store the data to shared memory.
  if (lane == 0) reduce_smem[warp] = prod;
  __syncthreads(); // make sure the data is in shared memory.
  // the first warp compute the final sum.
  prod = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f;
  if (warp == 0) prod = warp_reduce_sum<NUM_WARPS>(prod);
  if (tid == 0) atomicAdd(y, prod);
}
