#include "hip/hip_runtime.h"
#include<iostream>

// 将二维数组的行列索引转成一维数组的行列索引，这样可以更高效访问数据
// row, col：二维数组实际的行列索引，ld表示该数组实际的列数
// 例：二维数组实际的行列索引为(1, 3)，即第二行第四个元素，二维数据的总列数 = 5
// 返回的一位数组形式的索引为: 1*5 + 3 = 8
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// 定义naive gemm的kernel函数
__global__ void naiveSgemm(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {
    
    // 当前thread在C矩阵中的row
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    // 当前thread在C矩阵中的col
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (m < M && n < N) {
        float psum = 0.0;
        // 告知编译器自动展开循环体，这样可以减少循环控制的开销（循环次数小的时候可以这么做）
        #pragma unroll
        // 取出A[row]和B[col]，然后逐个元素相乘累加，得到最终结果
        for (int k = 0; k < K; k++) {
            // a[OFFSET(m, k, K)]: 获取A[m][k]
            // b[OFFSET(k, n, N)]: 获取B[k][n]
            psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        }
        c[OFFSET(m, n, N)] = psum;
    }
}

const int BM = 32, BN = 32;
const int M = 512, N = 512, K = 512;
dim3 blockDim(BN, BM);
dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

int main(){

}