#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "error.cuh"
#include "../include/kernel.cuh"

// Layer Norm: x: NxK(K=128<1024), y': NxK, y'=x-mean(x)/std(x) each row
// mean(x) = sum(x)/K, 1/std(x) = rsqrtf( sum( (x-mean(x))^2 )/K ) each row
// grid(N*K/K), block(K<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g + b (g: scale, b: bias)
template<const int NUM_THREADS=128>
__global__ void layer_norm(float* x, float* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const float epsilon = 1e-5f;

  __shared__ float s_mean; // shared within block
  __shared__ float s_variance; // shared within block
  float value = (idx < N * K) ? x[idx] : 0.0f; // load once only
  float sum = block_reduce_sum<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / (float) K;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();
  float variance = (value - s_mean) * (value - s_mean);
  variance = block_reduce_sum<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / (float) K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  if (idx < N * K) y[idx] = ((value - s_mean) * s_variance) * g + b;
}

// Layer Norm Vec4: x: NxK(K=128<1024), y': NxK, y'=x-mean(x)/std(x) each row
// mean(x) = sum(x)/K, 1/std(x) = rsqrtf( sum( (x-mean(x))^2 )/K ) each row
// grid(N*K/K), block(K/4<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g + b (g: scale, b: bias)
template<const int NUM_THREADS=128/4>
__global__ void layer_norm_vec4(float* x, float* y, float g, float b, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 4;
  const float epsilon = 1e-5f;

  __shared__ float s_mean; // shared within block
  __shared__ float s_variance; // shared within block
  float4 reg_x = FLOAT4(x[idx])
  float value = (idx < N * K) ? (reg_x.x + reg_x.y 
                               + reg_x.z + reg_x.w) : 0.0f;
  float sum = block_reduce_sum<NUM_THREADS>(value);
  if (tid == 0) s_mean = sum / (float) K;
  // wait for s_mean in shared memory to be ready for all threads
  __syncthreads();
  float4 reg_x_hat;
  reg_x_hat.x = reg_x.x - s_mean;
  reg_x_hat.y = reg_x.y - s_mean;
  reg_x_hat.z = reg_x.z - s_mean;
  reg_x_hat.w = reg_x.w - s_mean;
  float variance = reg_x_hat.x * reg_x_hat.x + reg_x_hat.y * reg_x_hat.y 
                 + reg_x_hat.z * reg_x_hat.z + reg_x_hat.w * reg_x_hat.w;
  variance = block_reduce_sum<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / (float) K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  float4 reg_y;
  reg_y.x = reg_x_hat.x * s_variance * g + b;
  reg_y.y = reg_x_hat.y * s_variance * g + b;
  reg_y.z = reg_x_hat.z * s_variance * g + b;
  reg_y.w = reg_x_hat.w * s_variance * g + b;
  if (idx < N * K) FLOAT4(y[idx]) = reg_y;
}
