#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "error.cuh"
#include "../include/kernel.cuh"


const double error = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;


void __global__ hello(){
    const int bx = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    printf("hello from %d bid and (%d, %d)!\n", bx, tx, ty);
}


void __global__ add(const double *d_x, const double *d_y, double *d_z, int64_t N) {
    const int tid = blockDim.x * blockIdx.x  + threadIdx.x;
    if (tid < N) {
        d_z[tid] = d_x[tid] + d_y[tid];
    }
}


// ElementWise Add  
// grid(N/128), block(128)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add(float* a, float* b, float* c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) c[idx] = a[idx] + b[idx];
}


// ElementWise Add + Vec4
// grid(N/128), block(128/4)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_vec4(float* a, float* b, float* c, int N) {
  int idx = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    float4 reg_a = FLOAT4(a[idx]);
    float4 reg_b = FLOAT4(b[idx]);
    float4 reg_c;
    reg_c.x = reg_a.x + reg_b.x;
    reg_c.y = reg_a.y + reg_b.y;
    reg_c.z = reg_a.z + reg_b.z;
    reg_c.w = reg_a.w + reg_b.w;
    FLOAT4(c[idx]) = reg_c;
  }
}


void check_acc(const double *z, const int64_t N) {
    bool has_error = false;
    for (int i=0; i<N; i++) {
        if (fabs(z[i]-c) > error){
            // printf("error is %f\n", z[i]-c);
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}


int main(){
    // dim3 grid_size = 2;
    // dim3 block_size(2,4);   
    // blockDim.x = 2, blockDim.y = 4
    // threadIdx.x -> (0, 1), threadIdx.y -> (0, 3)

    // NOTE: blockIdx.x * blockIdx.y * blockIdx.z <= 1024
    // hello<<<grid_size, block_size>>>();

    const int64_t N = 16;
    const int64_t M = sizeof(double)*N;
    double *h_x = (double*)malloc(M);
    double *h_y = (double*)malloc(M);
    double *h_z = (double*)malloc(M);

    for (int i=0; i<N; i++){
        h_x[i] = a;
        h_y[i] = b;
    }

    // alloc the memo for device
    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);

    // copy the data from host to device
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

    const int block_size = 4;
    const int grid_size = N / 4;

    // execution the add compute
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    // copy the result from host to device
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check_acc(h_z, N);
    
    free(h_x);
    free(h_y);
    free(h_z);
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));


    // hipDeviceSynchronize();
    return 0;
}