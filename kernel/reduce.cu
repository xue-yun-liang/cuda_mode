#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#include <float.h>

#ifdef DUSE_DP
typedef double real;
#else
typedef float real;
#endif

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

const int N=2048; // Example size, should be a power of 2 for simplicity
const int BLOCK_SIZE=128; // Example block size, can be tuned


// version 0: Reduce in CPU
real reduce_cpu(const real *x, const int N) {
  float sum = 0.0;
  for (int i = 0; i < N; i++)
  {
    sum += x[i];
  }
  return sum;
}

// version 1: a incorrect reduce in gpu, as
__global__ void recude_error(real *d_x, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int offset = N/2; offset>0; offset /=2){
    if (tid < offset) d_x[tid] += d_x[tid + offset];
  }
}

// version 2: a reduce using global memory
__global__ void reduce_global(real *d_x, real *d_y) {
  const int tid = threadIdx.x;
  real *x = d_x + blockIdx.x * blockDim.x;
  for (int offset=blockDim.x >> 1; offset>0; offset>>=1) {
    if(tid < offset) x[tid] += x[tid + offset];
    __syncthreads();
  }

  if (tid == 0) {
    d_y[blockIdx.x] = x[0];
  }
}


// version 3: a reduce using static shared memory
__global__ void reduce_share(real* d_x, real *d_y) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int n = bid * blockDim.x + tid;
  __shared__ real s_y[128];
  s_y[tid] = (n < N) ? d_x[n] : 0.0;
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) s_y[tid] = s_y[tid + offset];
    __syncthreads();
  }

  if (tid == 0) {
    d_y[tid] = s_y[0];
  }
}

// version 4: a reduce using dynmic shared memory
__global__ void reduce_share_(const real* d_x, real *d_y) {
  // call reduce_share_<<<grid_size, block_size, sizeof(real) * block>>>(x, y);
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int n = bid * blockDim.x + tid;
  extern __shared__ real s_y[];
  s_y[tid] = (n < N) ? d_x[n] : 0.0;
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) s_y[tid] = s_y[tid + offset];
    __syncthreads();
  }

  if (tid == 0) {
    d_y[tid] = s_y[0];
  }
}


// version 5: a reduce using atomic func
__global__ void reduce_share_atomic(const real* d_x, real *d_y, int N) {
  // call reduce_share_<<<grid_size, block_size, sizeof(real) * block>>>(x, y);
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int n = bid * blockDim.x + tid;
  extern __shared__ real s_y[];
  s_y[tid] = (n < N) ? d_x[n] : 0.0;
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) s_y[tid] = s_y[tid + offset];
    __syncthreads();
  }

  if (tid == 0) {
    atomicAdd(&d_y[0], s_y[0]);
  }
}

real ruNreduce(const real *d_x){
  const int grid_size = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
  const int smem = sizeof(real) * BLOCK_SIZE;

  real h_y[1] = {0};
  real *d_y;

  CHECK(hipMalloc(&d_y, sizeof(real)));
  CHECK(hipMemcpy(h_y, d_y, sizeof(real),hipMemcpyHostToDevice));

  reduce_share_atomic<<<grid_size, BLOCK_SIZE, smem>>>(d_x, d_y, N);

  CHECK(hipMemcpy(d_y, h_y, sizeof(real), hipMemcpyDeviceToHost));
  CHECK(hipFree(d_y));

  return h_y[0];
}


// version 6: reduce using __syncwarp
__global__ void reduce_syncwarp(const real *d_x, real *d_y, const int N){
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int n = bid * blockDim.x + tid;
  extern __shared__ real s_y[];
  s_y[tid] = (n < N) ? d_x[n] : 0.0;
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1) {
    if (tid < offset){
      s_y[tid] += s_y[tid + offset];
    }
    __syncthreads();
  }


  for (int offset = 16; offset > 0; offset >>= 1) {
    if (tid < offset){
      s_y[tid] += s_y[offset + tid];
    }
    __syncwarp();
  }

  if(tid == 0){
    atomicAdd(d_y, s_y[0]);
  }
}


// version 6: reduce using __shfl_xor_sync
__global__ void reduce_shfl_xor_sync(const real *d_x, real *d_y, const int N){
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int n = bid * blockDim.x + tid;
  extern __shared__ real s_y[];
  s_y[tid] = (n < N) ? d_x[n] : 0.0;
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1) {
    if (tid < offset){
      s_y[tid] += s_y[tid + offset];
    }
    __syncthreads();
  }

  real y = s_y[tid];

  for (int offset = 16; offset > 0; offset >>= 1) {
    y += __shfl_xor_sync(uint(-1), y, offset);
  }

  if(tid == 0){
    atomicAdd(d_y, y);
  }
}


void launch_reduce(const real *d_x, real *d_y, const int N) {
  const int block_size = 256;
  const int grid_size = (N + block_size - 1) / block_size;
  size_t shared_mem_size = block_size * sizeof(real);
  reduce_shfl_xor_sync<<<grid_size, block_size, shared_mem_size>>>(d_x, d_y, N);
}


real run_reduce(const real *h_x, real *h_y, const int N){
  const int block_size = 256;
  const int grid_size = (N + block_size - 1) / block_size;
  size_t shared_mem_size = block_size * sizeof(real);

  real *d_x, *d_y;
  hipMalloc(&d_x, N * sizeof(real));
  hipMalloc(&d_y, sizeof(real));

  hipMemcpy(d_x, h_x, N * sizeof(real), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, sizeof(real), hipMemcpyHostToDevice);

  reduce_shfl_xor_sync<<<grid_size, block_size, shared_mem_size>>>(d_x, d_y, N);

  hipMemcpy(h_y, d_y, sizeof(real), hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);
  return *h_y;
}


// Warp Reduce Sum
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Warp Reduce Max
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_max(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val = fmaxf(val, __shfl_xor_sync(0xffffffff, val, mask));
  }
  return val;
}

// Block reduce sum/max/min device helper for Layer/RMS Norm/Softmax etc.
// grid 1D block 1D, grid(N/128), block(128)
template<const int NUM_THREADS=128>
__device__ __forceinline__ float block_reduce_sum(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  val = warp_reduce_sum<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val = warp_reduce_sum<NUM_WARPS>(val);
  return val;
}

template<const int NUM_THREADS=128>
__device__ __forceinline__ float block_reduce_max(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  val = warp_reduce_max<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : -FLT_MAX;
  val = warp_reduce_max<NUM_WARPS>(val);
  return val;
}

// Kernel to test block reduce sum and max
__global__ void test_reduce_kernels() {
  float val = static_cast<float>(threadIdx.x);
  
  float sum = block_reduce_sum<128>(val);
  float max = block_reduce_max<128>(val);
  
  if (threadIdx.x == 0) {
    printf("Block reduce sum: %f\n", sum);
    printf("Block reduce max: %f\n", max);
  }
}

// int main() {
//   // Launch kernel with 1 block of 128 threads
//   test_reduce_kernels<<<1, 128>>>();
//   hipDeviceSynchronize();
  
//   return 0;
// }

void reduce_wrapper(real *d_x, real *d_y, int size) {
    int gridSize = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    reduce_global<<<gridSize, BLOCK_SIZE>>>(d_x, d_y);
}

int main() {
    real *h_x = new real[N];
    real h_y = 0;

    // Initialize input data
    for (int i = 0; i < N; i++) {
        h_x[i] = static_cast<real>(i + 1); // Example data
    }

    real res = run_reduce(h_x, &h_y, N);
    printf("%f\n",res);

    return 0;
}