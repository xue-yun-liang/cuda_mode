#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])


// Warp Reduce Sum
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Warp Reduce Max
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_max(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val = fmaxf(val, __shfl_xor_sync(0xffffffff, val, mask));
  }
  return val;
}

// Block reduce sum/max/min device helper for Layer/RMS Norm/Softmax etc.
// grid 1D block 1D, grid(N/128), block(128)
template<const int NUM_THREADS=128>
__device__ __forceinline__ float block_reduce_sum(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  val = warp_reduce_sum<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val = warp_reduce_sum<NUM_WARPS>(val);
  return val;
}

template<const int NUM_THREADS=128>
__device__ __forceinline__ float block_reduce_max(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  val = warp_reduce_max<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : -FLT_MAX;
  val = warp_reduce_max<NUM_WARPS>(val);
  return val;
}