// #include <stdio.h>
// #include <stdlib.h>
#include <iostream>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// SGEMM: Block Tile + K Tile, with smem
// Block Tile (BM, BN) + K Tile (BK=32)
// grid((N + BN - 1) / BN, (M + BM - 1) / BM), block(BN, BM)
// a: MxK, b: KxN, c: MxN, compute: c = a * b, all row major  
__global__ void sgemm(float* a, float* b, float* c, int M, int N, int K) {
  // [1] Block Tile: 32x32的block处理c上一块32x32的元素计算
  // [2]     K Tile: 使用共享内存，并将K分块为BK大小的块
  constexpr int BM = 32;
  constexpr int BN = 32;
  constexpr int BK = 32;
  __shared__ float s_a[BM][BK], s_b[BK][BN]; 

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tid = ty * blockDim.x + tx; // tid within the block
  // load values to shared memory, 32x32 threads working together 
  // to fetch data along the row direction of a and b both for s_a 
  // and s_b 32x32x4x2=8KB, we use 32x32 threads within block to 
  // load 32x32 elements from global memory to shared memory, namely, 
  // each thread will load 1 element.
  int load_smem_a_m = tid / 32; // 0~31, tid / 32, tid / BM, threadIdx.y
  int load_smem_a_k = tid % 32; // 0~31, tid % 32, tid % BK, threadIdx.x
  int load_smem_b_k = tid / 32; // 0~31, tid / 32, tid / BK, threadIdx.y
  int load_smem_b_n = tid % 32; // 0~31, tid % 32, tid % BN, threadIdx.x
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c
  // if (load_gmem_a_m >= M || load_gmem_b_n >= N) return;
  
  float sum = 0.f;
  for (int bk = 0; bk < (K + BK - 1) / BK; ++bk) {
    int load_gmem_a_k = bk * BK + load_smem_a_k;
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    s_a[load_smem_a_m][load_smem_a_k] = a[load_gmem_a_addr];
    int load_gmem_b_k = bk * BK + load_smem_b_k;
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;
    s_b[load_smem_b_k][load_smem_b_n] = b[load_gmem_b_addr];
    __syncthreads();
    #pragma unroll
    for (int k = 0; k < BK; ++k) {
      int comp_smem_a_m = load_smem_a_m;
      int comp_smem_b_n = load_smem_b_n;
      sum += s_a[comp_smem_a_m][k] * s_b[k][comp_smem_b_n];
    }
    __syncthreads();
  }
  int store_gmem_c_m = load_gmem_a_m;
  int store_gmem_c_n = load_gmem_b_n;
  int store_gmem_c_addr = store_gmem_c_m * N + store_gmem_c_n;
  c[store_gmem_c_addr] = sum;
}

// SGEMM: Block Tile + Thread Tile + K Tile + Vec4, with smem
// BK:TILE_K=8 BM=BN=128
// TM=TN=8 增加计算密度 BM/TM=16 BN/TN=16
// dim3 blockDim(BN/TN, BM/TM);
// dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM)
__global__ void sgemm_thread_tile_vec4(
  float* a, float* b, float* c, int M, int N, int K) {
  // [1]  Block Tile: 一个16x16的block处理C上大小为128X128的一个目标块
  // [2] Thread Tile: 每个thread负责计算TM*TN(8*8)个元素，增加计算密度
  // [3]      K Tile: 将K分块，每块BK大小，迭代(K+BK-1/BK)次，
  //                  每次计算TM*TN个元素各自的部分乘累加
  // [4]   Vectorize: 减少load和store指令，使用float4
  constexpr int BM = 128;
  constexpr int BN = 128;
  constexpr int BK = 8; 
  constexpr int TM = 8;
  constexpr int TN = 8;

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tid = ty * blockDim.x + tx; // tid within the block
  __shared__ float s_a[BM][BK], s_b[BK][BN]; // 2*128*8*4=8KB
  
  // 0. 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=8 按行读取 A行主序
  // 对于s_a每行8个数据，每个线程读取4个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2; // tid/2 (128/8)*(128/8)=256 threads per block, tid/2->[0,128), BM=128 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 4;  // (tid%2 == 0) ? 0 : 4, col of s_a 0,4
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=8 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读4个数据，需要32个线程；总共8行，需要32x8=256个线程
  int load_smem_b_k = tid / 32; // tid/32, row of s_b 256/32=8 行 0~7
  int load_smem_b_n = (tid % 32) * 4;  // (tid % 32) * 4, col of s_b 0,4,...,124
  // 1. 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c
  
  float r_c[TM][TN] = {0.0}; // 8x8
  // 2. 先对K进行分块，每块BK大小
  for (int bk = 0; bk < (K + BK - 1) / BK; ++bk) {
    // 加载数据到共享内存smem s_a BM*BK 128*8 vectorize float4
    int load_gmem_a_k = bk * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    FLOAT4(s_a[load_smem_a_m][load_smem_a_k]) = FLOAT4(a[load_gmem_a_addr]);
    // 加载数据到共享内存smem s_b BK*BN 8*128 vectorize float4
    int load_gmem_b_k = bk * BK + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 
    FLOAT4(s_b[load_smem_b_k][load_smem_b_n]) = FLOAT4(b[load_gmem_b_addr]); 
    __syncthreads();
    #pragma unroll
    for (int k = 0; k < BK; k++) {
      // 3. 每个线程负责计算BM*BN(12x128)中的TM*TN(8x8)个元素
      #pragma unroll
      for (int m = 0; m < TM; m++) {
        #pragma unroll
        for (int n = 0; n < TN; n++) {
          // k from 0~7，0 ~ BK, ty and tx range from 0 to 15, 16x8=128
          int comp_smem_a_m = ty * TM + m;  // 128*8 128/TM(8)=16 M方向 16线程
          int comp_smem_b_n = tx * TN + n;  // 8*128 128/TN(8)=16 N方向 16线程
          r_c[m][n] += s_a[comp_smem_a_m][k] * s_b[k][comp_smem_b_n];
        }
      }
    }
    __syncthreads();
  }

  #pragma unroll
  for (int m = 0; m < TM; ++m) {
    int store_gmem_c_m = by * BM + ty * TM + m;
    #pragma unroll
    for (int n = 0; n < TN; n += 4) {
      int store_gmem_c_n = bx * BN + tx * TN + n;
      int store_gmem_c_addr = store_gmem_c_m * N + store_gmem_c_n;
      FLOAT4(c[store_gmem_c_addr]) = FLOAT4(r_c[m][n]);
    }
  }
}

void launch_sgemm(float* h_a, float* h_b, float* h_c, int M, int N, int K){
  dim3 block(32, 32);
  dim3 grid((N + 31) / 32, (M + 31) / 32);

  // Launch the kernel
  sgemm<<<grid, block>>>(h_a, h_b, h_c, M, N, K);
}

void launch_sgemm_thread_tile_vec4(float* h_a, float* h_b, float* h_c, int M, int N, int K){
  dim3 block(32, 32);
  dim3 grid((N + 31) / 32, (M + 31) / 32);

  // Launch the kernel
  sgemm_thread_tile_vec4<<<grid, block>>>(h_a, h_b, h_c, M, N, K);
}

void run_sgemm(float* h_a, float* h_b, float* h_c, int M, int N, int K) {
    float *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void**)&d_a, M * K * sizeof(float));
    hipMalloc((void**)&d_b, K * N * sizeof(float));
    hipMalloc((void**)&d_c, M * N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_a, h_a, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, K * N * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 block(32, 32);
    dim3 grid((N + 31) / 32, (M + 31) / 32);

    // Launch the kernel
    sgemm<<<grid, block>>>(d_a, d_b, d_c, M, N, K);

    // Copy the result back to the host
    hipMemcpy(h_c, d_c, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main() {
    int M = 64;
    int N = 64;
    int K = 64;

    float h_a[M * K];
    float h_b[K * N];
    float h_c[M * N];

    // Initialize matrices h_a and h_b with some values
    for (int i = 0; i < M * K; ++i) {
        h_a[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < K * N; ++i) {
        h_b[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Run SGEMM
    run_sgemm(h_a, h_b, h_c, M, N, K);

    // Optionally: Print the result matrix h_c

    return 0;
}